#include "cuda_handler.hpp"
#include <stdexcept>

namespace msvd {

// Singleton instance initialization
CUDAHandler* CUDAHandler::instance = nullptr;

CUDAHandler* CUDAHandler::get_instance() {
   if (instance == nullptr) {
      instance = new CUDAHandler();
   }
   return instance;
}

void CUDAHandler::init() {
   CUDAHandler* handler = get_instance();
   
   CUBLAS_CHECK(hipblasCreate(&handler->cublas_handle));
   CUSPARSE_CHECK(hipsparseCreate(&handler->cusparse_handle));
   CUSOLVER_CHECK(hipsolverDnCreate(&handler->cusolver_handle));
}

void CUDAHandler::finalize() {
   if (instance != nullptr) {
      CUBLAS_CHECK(hipblasDestroy(instance->cublas_handle));
      CUSPARSE_CHECK(hipsparseDestroy(instance->cusparse_handle));
      CUSOLVER_CHECK(hipsolverDnDestroy(instance->cusolver_handle));
      
      delete instance;
      instance = nullptr;
   }
}

hipblasHandle_t CUDAHandler::cublas() {
   return get_instance()->cublas_handle;
}

hipsparseHandle_t CUDAHandler::cusparse() {
   return get_instance()->cusparse_handle;
}

hipsolverHandle_t CUDAHandler::cusolver() {
   return get_instance()->cusolver_handle;
}

} // namespace msvd 