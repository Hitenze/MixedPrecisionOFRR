#include "hip/hip_runtime.h"
#include "rr_svd.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <stdexcept>
#include <algorithm>
#include <cmath>

namespace msvd {

template<typename T_I, typename T_O, typename T_COMPUTE>
MSVDStatus rr_svd(const Matrix<T_I>& A, Matrix<T_O>& U, Matrix<T_O>& S, Matrix<T_O>& V, 
             bool use_generalized, T_COMPUTE select_tol) {
   
   return MSVDStatus::kSuccess;
}

// Explicit instantiation for supported types
template MSVDStatus rr_svd<double, double, double>(const Matrix<double>& A, Matrix<double>& U, Matrix<double>& S, Matrix<double>& V, bool use_generalized, double select_tol);
template MSVDStatus rr_svd<float, float, float>(const Matrix<float>& A, Matrix<float>& U, Matrix<float>& S, Matrix<float>& V, bool use_generalized, float select_tol);
template MSVDStatus rr_svd<double, float, float>(const Matrix<double>& A, Matrix<float>& U, Matrix<float>& S, Matrix<float>& V, bool use_generalized, float select_tol);
template MSVDStatus rr_svd<float, double, double>(const Matrix<float>& A, Matrix<double>& U, Matrix<double>& S, Matrix<double>& V, bool use_generalized, double select_tol);
template MSVDStatus rr_svd<__half, __half, float>(const Matrix<__half>& A, Matrix<__half>& U, Matrix<__half>& S, Matrix<__half>& V, bool use_generalized, float select_tol);
template MSVDStatus rr_svd<__half, float, float>(const Matrix<__half>& A, Matrix<float>& U, Matrix<float>& S, Matrix<float>& V, bool use_generalized, float select_tol);

} // namespace msvd 